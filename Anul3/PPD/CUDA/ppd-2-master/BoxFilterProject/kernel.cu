﻿
#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <fstream>


const int n=3,m=3;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;


__constant__ const float W[3][3] = {
    1.0 / 9 , 1.0 / 9 , 1.0 / 9 ,
    1.0 / 9 , 1.0 / 9 , 1.0 / 9 ,
    1.0 / 9 , 1.0 / 9 , 1.0 / 9 
};


hipError_t boxFilterCuda(Matrix& IMG_IN, Matrix& IMG_OUT);

__global__ void boxFilter(Matrix IMG_IN, Matrix IMG_OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; //row
    int j = blockIdx.y * blockDim.y + threadIdx.y; //col

    int N = IMG_IN.width, M = IMG_IN.height;


    for (int k = 0; k < n; ++k)
        for (int l = 0; l < m; ++l)
            if (i - n / 2 + k >= 0 && i - n / 2 + k < N && j - m / 2 + l >= 0 && j - m / 2 + l < M)
            {
                IMG_OUT.elements[i * N + j] += (IMG_IN.elements[(i - n / 2 + k)*N + j - m / 2 + l] * W[k][l]);
               // printf("%f", IMG_OUT.elements[(i - n / 2 + k) * N + j - m / 2 + l]);
            }



}
void readImage(const std::string& fName, Matrix& IMG_IN)
{
    std::ifstream fin(fName);
    int height, width;
    fin >> height >> width;
    for (int i = 0; i < height; ++i)
        for (int j = 0; j < width; ++j)
            fin >> IMG_IN.elements[i * width + j];

    
}

// Thread block size
#define BLOCK_SIZE 10

int main()
{
    const int N = 10;
    const int M = 10;
    Matrix IMG_IN;
    Matrix IMG_OUT;

    IMG_IN.elements = new float[N * M];
    IMG_IN.height = N;
    IMG_IN.width = M;

    IMG_OUT.elements = new float[N * M];
    IMG_OUT.height = N;
    IMG_OUT.width = M;

    for (int i = 0; i < 10; ++i)
        for (int j = 0; j < 10; ++j)
            IMG_IN.elements[i*N + j] = i * 10 + j;

    readImage("in.txt", IMG_IN);

    hipError_t cudaStatus = boxFilterCuda(IMG_IN, IMG_OUT);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "BOX FILTER FAILED!");
        return 1;
    }
    

    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; ++j) {
            printf("%.6f ", IMG_OUT.elements[i * N + j]);
        }
        printf("\n");
    }


    return 0;
}

hipError_t boxFilterCuda(Matrix& h_in, Matrix& h_out)
{
    Matrix dev_in;
    dev_in.height = h_in.height;
    dev_in.width = h_in.width;
    size_t size = dev_in.width * dev_in.height * sizeof(int);
    hipMalloc(&dev_in.elements, size);
    hipMemcpy(dev_in.elements, h_in.elements, size, hipMemcpyHostToDevice);


    Matrix dev_out;
    dev_out.height = h_in.height;
    dev_out.width = h_in.width;
    hipMalloc(&dev_out.elements, size);

    hipError_t cudaStatus;

    // blocksize
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // gridsize
    dim3 dimGrid(h_in.width / dimBlock.x, h_in.height / dimBlock.y);
    
    // Launch a kernel on the GPU with one thread for each element.
    boxFilter <<< dimGrid, dimBlock >>> (dev_in, dev_out);
   
    // Read C from device memory
    cudaStatus = hipMemcpy(h_out.elements, dev_out.elements, size,
        hipMemcpyDeviceToHost);


    
    hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemCpy failed!");
        goto Error;
    }



Error:
    hipFree(dev_in.elements);
    hipFree(dev_out.elements);

    return cudaStatus;
}
